#include "hip/hip_runtime.h"
#include "stencil.h"

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK
#define N_NEIGHBOURS_2 2

using namespace std;

__global__ 
void stencilKernel (float *a, float *c) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// **************
	// to be used only on exercise 2
	 __shared__ float temp[NUM_THREADS_PER_BLOCK + 2 * N_NEIGHBOURS_2];
	// now, fill the temp array
	 temp[threadIdx.x +  N_NEIGHBOURS_2] = a[id];
	if (threadIdx.x == 0){
		for(int i = 0 , i < N_NEIGHBOURS_2; i++)
			temp[threadIdx.x - i] = a[id -i]; 
	}
	 __syncthreads();
	// **************

	// initialise the array with the results
	c[id] = 0;

	// iterate through the neighbours required to calculate
	// the values for the current position of c
	for (int i = threadIdx.x ; i < threadIdx.x + 2 * N_NEIGHBOURS_2 ; i++) {
		c[id] += temp[i];
	}

}

void stencil (float *a, float *c) {
	chrono::steady_clock::time_point begin = chrono::steady_clock::now();

	for (int i = 0; i < SIZE; i++) {
		// considers 4 neighbours
		for (int n = -2; n <= 2; n++) {
			if ((i + n >= 0) && (i + n < SIZE))
				c[i] += a[i + n];
		}
	}

	chrono::steady_clock::time_point end = chrono::steady_clock::now();
	cout << endl << "Sequential CPU execution: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl << endl;
}

void launchStencilKernel (float *a, float *c) {
	// pointers to the device memory
	float *da, *dc;
	// declare variable with size of the array in bytes
	int bytes = SIZE * sizeof(float);

	// allocate the memory on the device
	hipMalloc ((void**) &da, bytes);
	hipMalloc ((void**) &dc, bytes);
	checkCUDAError("mem allocation");

	// copy inputs to the device
	hipMemcpy (da, a, bytes, hipMemcpyHostToDevice);
	checkCUDAError("memcpy h->d");

	// launch the kernel
	startKernelTime ();
	stencilKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (da, dc);
	stopKernelTime ();
	checkCUDAError("kernel invocation");

	// copy the output to the host
	hipMemcpy (c, dc, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");

	// free the device memory
	hipFree(da);
	hipFree(dc);
	checkCUDAError("mem free");
}

int main( int argc, char** argv) {
	// arrays on the host
	float a[SIZE], b[SIZE], c[SIZE];

	// initialises the array
	for (unsigned i = 0; i < SIZE; ++i)
		a[i] = (float) rand() / RAND_MAX;

	stencil (a, b);
	
	launchStencilKernel (a, c);

	return 0;
}
